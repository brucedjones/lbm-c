#include "hip/hip_runtime.h"
#ifndef KERNEL
#define KERNEL

#ifdef _WIN64
	#pragma comment(lib, "cgns/x64/lib/cgns.lib")
	#include "cgns\x64\include\cgnslib.h"
	#pragma comment(lib, "HDF5/x64/lib/hdf5.lib")
	#include "HDF5/x64/include/hdf5.h"
	#pragma comment(lib, "HDF5/x64/lib/libszip.lib")
	#include "HDF5/x64/include/szlib.h"
	#pragma comment(lib, "HDF5/x64/lib/libzlib.lib")
	#include "HDF5/x64/include/zlib.h"
#else
	#pragma comment(lib, "cgns/x86/lib/cgns.lib")
	#include "cgns\x86\include\cgnslib.h"
	#pragma comment(lib, "HDF5/x86/lib/hdf5.lib")
	#include "HDF5/x86/include/hdf5.h"
	#pragma comment(lib, "HDF5/x86/lib/libszip.lib")
	#include "HDF5/x86/include/szlib.h"
	#pragma comment(lib, "HDF5/x86/lib/libzlib.lib")
	#include "HDF5/x86/include/zlib.h"
#endif

#include <stdio.h>
#include "data_types.cuh"
#include "macros.cu"
#include "solver.cu"
#include "index.cuh"
#include "model_builder.cu"
#include "cgns/cgns_output_handler.cu"
#include "cuda_util.cu"

// Include THRUST libraries
#include <thrust/transform_reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/device_vector.h>

// DEVICE VARIABLE DECLARATION
Lattice *lattice_device;
Domain *domain_device;
DomainConstant *domain_constants_device;
OutputController *output_controller_device;

// HOST VARIABLE DECLARATION
Lattice *lattice_host, *lattice_device_prototype;
Domain *domain_host;
DomainConstant *domain_constants_host;
OutputController *output_controller_host;
Timing *times;
ProjectStrings *project;
ModelBuilder model_builder;


// SCALAR DECLARATION (PLATFORM AGNOSTIC)
bool store_macros = false;

// DECLARE OUTPUT HANDLER
CGNSOutputHandler output_handler;

int main(int argc, char **argv)
{

	// Get available memory on graphics card before allocation
	size_t freeMemory_before = 0;
	size_t totalMemory_before = 0;
	hipMemGetInfo(&freeMemory_before, &totalMemory_before);
	
	// Initialise memory for LBM model
	setup(argv[1]);
	
	// Get available memory on graphics card after allocation
	size_t freeMemory_after = 0;
	size_t totalMemory_after = 0;
	hipMemGetInfo(&freeMemory_after, &totalMemory_after);

	// Report program memory usage
	cout << "Total Device Memory:	 "<< totalMemory_after / 1024 / 1024 << "Mb" << endl;
	cout << "Total Availabe Memory:	 "<< freeMemory_before / 1024 / 1024 << "Mb" << endl;
	cout << "Memory Used:            "<< (freeMemory_before-freeMemory_after) / 1024 / 1024 << "Mb" << endl;

	// Report domain configuration
	printf("X-Length:		%d\n", domain_constants_host->length[0]);
	printf("Y-Length:		%d\n", domain_constants_host->length[1]);
	#if DIM > 2
		printf("Z-Length:		%d\n", domain_constants_host->length[2]);
	#endif
	printf("Relaxation Time (Tau):	%f\n", domain_constants_host->tau);
	printf("\nPress return to continue...");
	if (output_controller_host->interactive == true) getchar();

	// Get current clock cycle number
	clock_t t1=clock();

	int domain_size=1;
	int stop=0;
	for(int d = 0; d<DIM ;d++)
	{
		domain_size = domain_size*domain_constants_host->length[d];
	}

	for(int i = 1; i<times->max+1; i++)
	{
		if((times->plot>0 && i%times->plot == 0) ||
		   (times->steady_check>0 && i%times->steady_check) || 
		   (times->screen>0 && i%times->screen)) store_macros = true;

		iterate(i-1);

		if(times->plot>0 && i%times->plot == 0)
		{
			output_macros(i);
			store_macros = false;
		}

		if(times->screen>0 && i%times->screen == 0)
		{
			screen_mess(i,output_controller_host->screen_node);
			store_macros = false;
		}

		if(times->steady_check>0 && i%times->steady_check == 0)
		{
			compute_residual(i);
			
			for(int resid=0;resid<NUM_RESIDS;resid++)
			{
				if(domain_constants_host->residual[resid]<domain_constants_host->tolerance) stop += 1;
			}
			if(isIndeterminate(domain_constants_host->residual[i%NUM_RESIDS]))
			{
				output_macros(i);
				exit(1);
			} else if(stop==NUM_RESIDS)
			{
				output_macros(i);
				break;
			}
			stop = 0;
			store_macros = false;
		}
	}

	// Get current clock cycle number
	clock_t t2=clock();
	// Compare and report global execution time
	double cputime = ((double)t2-(double)t1)/(double)CLOCKS_PER_SEC;
	printf("\n\nTotal Run Time: %fs",cputime);
	printf("\nPress return to finish");
	if (output_controller_host->interactive == true) getchar();


}


// EXECUTES ALL ROUTINES REQUIRED FOR THE MODEL SET UP
void setup(char *data_file)
{
	// Set cuda device to use
	hipSetDevice(0);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(iterate_kernel), hipFuncCachePreferL1);
	
	// Allocate container structures
	combi_malloc<Lattice>(&lattice_host, &lattice_device, sizeof(Lattice));
	combi_malloc<Domain>(&domain_host, &domain_device, sizeof(Domain));
	combi_malloc<DomainConstant>(&domain_constants_host, &domain_constants_device, sizeof(DomainConstant));
	combi_malloc<OutputController>(&output_controller_host, &output_controller_device, sizeof(OutputController));
	domain_constants_host = (DomainConstant *)malloc(sizeof(DomainConstant));
	times = (Timing *)malloc(sizeof(Timing));
	project = (ProjectStrings *)malloc(sizeof(ProjectStrings));
	lattice_device_prototype = (Lattice *)malloc(sizeof(Lattice));

	ModelBuilder tmpmb(data_file, lattice_host, lattice_device,
		domain_constants_host, domain_constants_device,
		domain_host, domain_device,
		output_controller_host, output_controller_device,
		times, project);
	model_builder = tmpmb;

	int z_len = 1;
	#if DIM > 2
		z_len = domain_constants_host->length[2];
	#endif
	CGNSOutputHandler tmp(project->output_fname,domain_constants_host->length[0],domain_constants_host->length[1],z_len);
	output_handler = tmp;
}



// COPIES f_i DATA FROM DEVICE TO HOST AND COMPUTERS MACROSCOPIC VALUES ON HOST, THIS DATA
// IS THEN WRITTEN TO THE OUTPUT FILE
//
// Note:	A computationally more efficient implementation would compute macroscopic
//			value's on the gpu and then just copy that data, this would however consume
//			more memory
void output_macros(int time)
{
	int domain_size = domain_constants_host->length[0]*domain_constants_host->length[1];
	#if DIM > 2
		domain_size = domain_size*domain_constants_host->length[2];
	#endif

	Domain domain_tmp;

	cudasafe(hipMemcpy(&domain_tmp, domain_device, sizeof(Domain),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");
	
	double *u_tmp[DIM];
	cudasafe(hipMemcpy(u_tmp, domain_tmp.u, sizeof(double*)*DIM,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	for(int d=0;d<DIM;d++)
	{
		cudasafe(hipMemcpy(domain_host->u[d], u_tmp[d], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");
	}

	double *rho_tmp;
	cudasafe(hipMemcpy(domain_host->rho, domain_tmp.rho, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	// Copy data from device to host
	//cudasafe(hipMemcpy(lattice_host->rho, lattice_device->rho, sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - rho");
	//cudasafe(hipMemcpy(lattice_host->u[0], lattice_device->u[0], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - ux");
	//cudasafe(hipMemcpy(lattice_host->u[1], lattice_device->u[1], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - uy");

	int num_fields = 0;
	if (output_controller_host->u[0] == true) num_fields++;
	if (output_controller_host->u[1] == true) num_fields++;
#if DIM > 2
	if (output_controller_host->u[2] == true) num_fields++;
#endif
	if (output_controller_host->rho == true) num_fields++;

	char **labels;
	double **data;

	labels = (char **)malloc(num_fields * sizeof (char *));
	data = (double **)malloc(num_fields * sizeof(double));

	for(int i = 0; i<num_fields;i++)
	{
		labels[i] = (char *)malloc(STR_LENGTH*sizeof(char));
	}

	int counter = 0;

	if (output_controller_host->u[0] == true)
	{
		data[counter] = domain_host->u[0];
		strcpy(labels[counter],"VelocityX");
		counter++;
	}

	if (output_controller_host->u[1] == true)
	{
		data[counter] = domain_host->u[1];
		strcpy(labels[counter],"VelocityY");
		counter++;
	}
#if DIM > 2
	if (output_controller_host->u[2] == true)
	{
		data[counter] = domain_host->u[2];
		strcpy(labels[counter],"VelocityZ");
		counter++;
	}
#endif	
	if (output_controller_host->rho == true)
	{
		data[counter] = domain_host->rho;
		strcpy(labels[counter],"Density");
		counter++;
	}

/*	data[0] = lattice_host->rho;
	data[1] = lattice_host->u[0];
	data[2] = lattice_host->u[1];

	strcpy(labels[0],"Density");
	strcpy(labels[1],"VelocityX");
	strcpy(labels[2],"VelocityY");*/

	output_handler.append_solution_output(time,num_fields,data,labels);
}

// CONFIGURES THE KERNEL CONFIGURATION AND LAUNCHES KERNEL
void iterate(int t)
{
	// GRID AND BLOCK DEFINITIONS CAN BE CALCULATED BEFORE ITERATE
	// DEFINE GRID AND BLOCK DIMS
	int3 threads;
	threads.x = (int)ceilf((float)domain_constants_host->length[0]/(float)NUM_THREADS_DIM_X);
	threads.y = (int)ceilf((float)domain_constants_host->length[1]/(float)NUM_THREADS_DIM_Y);
	threads.z = 1;

	int3 blocks;
	blocks.x = NUM_THREADS_DIM_X;
	blocks.y = NUM_THREADS_DIM_Y;
	blocks.z = 1;

	#if DIM >2
		threads.z = (int)ceilf((float)domain_constants_host->length[2]/(float)NUM_THREADS_DIM_Z);;
		blocks.z = NUM_THREADS_DIM_Z;
	#endif

	dim3 grid_dim = dim3(threads.x,threads.y,threads.z);
    dim3 block_dim = dim3(blocks.x,blocks.y,blocks.z);
	hipDeviceSynchronize();
	Check_CUDA_Error("Kernel \"iterate_bulk 1\" Execution Failed!");  
	// ITERATE ONCE
	iterate_kernel<<<grid_dim, block_dim>>>(lattice_device, domain_device, store_macros,t);
	hipDeviceSynchronize();
	Check_CUDA_Error("Kernel \"iterate_bulk 1\" Execution Failed!");  
	// SWAP CURR AND PREV LATTICE POINTERS READY FOR NEXT ITER
	//swap_lattices();
}

#if DIM > 2
	struct energy
	{
	    template <typename Tuple>
	    __host__ __device__
	    void operator()(Tuple t)
	    {
	        thrust::get<4>(t) = 0.5*thrust::get<3>(t)*((thrust::get<0>(t)*thrust::get<0>(t)) + (thrust::get<1>(t)*thrust::get<1>(t)) + (thrust::get<2>(t)*thrust::get<2>(t)));
	    }
	};
#else
	struct energy
	{
	    template <typename Tuple>
	    __host__ __device__
	    void operator()(Tuple t)
	    {
	        thrust::get<3>(t) = 0.5*thrust::get<2>(t)*((thrust::get<0>(t)*thrust::get<0>(t)) + (thrust::get<1>(t)*thrust::get<1>(t)));
	}
	};
#endif


double current_RMS(double *device_var_u[DIM], double *device_var_rho, int var_size)
{
	double *result;
	cudasafe(hipMalloc((void **)&result,sizeof(double)*var_size), "Model Builder: Device memory allocation failed!");

	// wrap raw pointer with a device_ptr for thrust compatibility
	thrust::device_ptr<double> dev_ptr_x(device_var_u[0]);
	thrust::device_ptr<double> dev_ptr_y(device_var_u[1]);
	#if DIM > 2
		thrust::device_ptr<double> dev_ptr_z(device_var_u[2]);
	#endif
	thrust::device_ptr<double> dev_ptr_rho(device_var_rho);
	thrust::device_ptr<double> dev_ptr_res(result);

	// apply the transformation
	#if DIM > 2
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(dev_ptr_x, dev_ptr_y, dev_ptr_z, dev_ptr_rho, dev_ptr_res)),
                     thrust::make_zip_iterator(thrust::make_tuple(dev_ptr_x+var_size, dev_ptr_y+var_size, dev_ptr_z+var_size, dev_ptr_rho+var_size, dev_ptr_res+var_size)),
                     energy());
	#else
		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(dev_ptr_x, dev_ptr_y, dev_ptr_rho, dev_ptr_res)),
                     thrust::make_zip_iterator(thrust::make_tuple(dev_ptr_x+var_size, dev_ptr_y+var_size, dev_ptr_rho+var_size, dev_ptr_res+var_size)),
                     energy());
	#endif
	Check_CUDA_Error("Steady State Calculation Kernel Execution Failed!");  
    
	// Compute RMS value
	//double sum = thrust::reduce(dev_ptr_res, dev_ptr_res+var_size, (double) 0, thrust::plus<double>());
	//double curr_RMS = sqrt(sum/var_size);

	double curr_RMS = thrust::reduce(dev_ptr_res, dev_ptr_res+var_size, (double) 0, thrust::plus<double>());

	cudasafe(hipFree(result),"Freeing Device Memory");

	return curr_RMS;
}

double prev_RMS = 0;

double error_RMS(double *device_var_u[DIM], double *device_var_rho, int var_size)
{
	double curr_RMS = current_RMS(device_var_u, device_var_rho, var_size);
	double tmp = ((abs(curr_RMS-prev_RMS)/times->steady_check))/curr_RMS;

	prev_RMS = curr_RMS;

	return tmp;
}

void compute_residual(int time)
{
	int domain_size = domain_constants_host->length[0]*domain_constants_host->length[1];
	#if DIM > 2
		domain_size = domain_size*domain_constants_host->length[2];
	#endif

	Domain domain_tmp;

	cudasafe(hipMemcpy(&domain_tmp, domain_device, sizeof(Domain),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	double *u_tmp[DIM];
	cudasafe(hipMemcpy(u_tmp, domain_tmp.u, sizeof(double*)*DIM,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	//double *rho_tmp;
	//cudasafe(hipMemcpy(rho_tmp, domain_tmp.rho, sizeof(double*),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	/*cudasafe(hipMemcpy(domain_host->u[0], u_tmp[0], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - u");
	cudasafe(hipMemcpy(domain_host->u[1], u_tmp[1], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - u");
	cudasafe(hipMemcpy(domain_host->u[2], u_tmp[2], sizeof(double)*domain_size,hipMemcpyDeviceToHost),"Copy Data: Output Data - u");*/

//	domain_constants_host->residual = error_RMS(u_tmp[0],u_tmp[1],u_tmp[2], rho_tmp,domain_size);
	domain_constants_host->residual[time%NUM_RESIDS] = error_RMS(u_tmp, domain_tmp.rho,domain_size);
}

void screen_mess(int iter, int coord[DIM])
{
	int idx = coord[0]+coord[1]*domain_constants_host->length[0];
	#if DIM > 2
		idx += coord[2]*domain_constants_host->length[0]*domain_constants_host->length[1];
	#endif

	double u[DIM],rho;
	Domain domain_tmp;

	cudasafe(hipMemcpy(&domain_tmp, domain_device, sizeof(Domain),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");
	
	double *u_tmp[DIM];
	cudasafe(hipMemcpy(u_tmp, domain_tmp.u, sizeof(double*)*DIM,hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	for(int d=0;d<DIM;d++)
	{
		cudasafe(hipMemcpy(&u[d], &u_tmp[d][idx], sizeof(double),hipMemcpyDeviceToHost),"Model Builder: BLAHBLAHCopy from device memory failed!");
	}

	cudasafe(hipMemcpy(&rho, &domain_tmp.rho[idx], sizeof(double),hipMemcpyDeviceToHost),"Model Builder: Copy from device memory failed!");

	cout << "time = " << iter << "; rho = " << rho << "; uX = " << u[0]<< "; uY = " << u[1] << "; ";
	#if DIM>2
		cout << "uZ = " << u[2] << "; ";
	#endif
	cout << "resid = " << domain_constants_host->residual[iter%NUM_RESIDS] << endl;
}

bool isIndeterminate(const double pV)
{
    return (pV != pV);
} 

#endif
